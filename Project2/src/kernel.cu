#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
//#include "glm/glm.hpp"
#include "utilities.h"
#include "kernel.h"


//#include "cuPrintf.cu"


//GLOBALS
dim3 threadsPerBlock(blockSize);

int numObjects;


void checkCUDAError(const char *msg, int line = -1)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        if( line >= 0 ) {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
        exit(EXIT_FAILURE); 
    }
} 


__global__ void naiveParallelScan (const int *a, int *b, int *temp, int size)
{
	int index = threadIdx.x;

	if(index < size){
		temp[index] = a[index];
		__syncthreads();

		for(int offset = 1; offset <= size; offset *= 2){

			if(index >= offset){
				b[index] = temp[index - offset] + temp[index];
			}
			else{
				b[index] = temp[index];
			}
			temp[index] = b[index];
			__syncthreads();
		}
	}
}


__global__ void sharedMemoryParallelScan (const int *a, int *b, int size){
	__shared__ int temp[blockSize];
	__shared__ int temp2[blockSize];

	int index = threadIdx.x;
	if(index < size )
	{
		temp[index] = a[index];
		__syncthreads();

		for(int offset = 1; offset <= size; offset *= 2){

			if(index >= offset){
				temp2[index] = temp[index - offset] + temp[index];
			}
			else{
				temp2[index] = temp[index];
			}
			__syncthreads();
		
			temp[index] = temp2[index];
		}
		b[index] = temp[index];
	}
}

__global__ void sharedMemoryParallelScanArbritraryLength (const int *a, int *b, int size, int offset){
	__shared__ int temp[blockSize];
	//__shared__ int temp2[blockSize];

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;


	temp[threadIdx.x] = a[index];
	__syncthreads();

	if(index < size){
		if(index >= offset){
			b[index] = a[index - offset] + temp[threadIdx.x];
		}
		else{
			b[index] = temp[threadIdx.x];
		}
		__syncthreads();
	}
}

__global__ void sharedMemoryParallelScanArbritraryLength2 (const int *a, int *b, int offset, int size){
	__shared__ int temp[blockSize];
	__shared__ int temp2[blockSize];

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	temp[threadIdx.x] = a[index];
	temp2[threadIdx.x] = a[index - offset];

	__syncthreads();



	if(index >= offset){
		b[index] = temp2[threadIdx.x] + temp[threadIdx.x];
	}
	else{
		b[index] = temp[threadIdx.x];
	}
	__syncthreads();

}

__global__ void scan(float *g_odata, float *g_idata, int n) 
{ 
	 extern __shared__ float temp[]; // allocated on invocation 
	 int thid = threadIdx.x; 
	 int pout = 0, pin = 1; 
	 // load input into shared memory. 
	 // This is exclusive scan, so shift right by one and set first elt to 0 
	 temp[pout*n + thid] = (thid > 0) ? g_idata[thid-1] : 0; 
	 __syncthreads(); 
	 for (int offset = 1; offset < n; offset *= 2) 
	 { 
		 pout = 1 - pout; // swap double buffer indices 
		 pin = 1 - pout; 
		 if (thid >= offset) 
			temp[pout*n+thid] += temp[pin*n+thid - offset]; 
		 else 
			temp[pout*n+thid] = temp[pin*n+thid]; 
		 __syncthreads(); 
	 } 
	 g_odata[thid] = temp[pout*n + thid]; // write output 
} 

__global__ void parallelScatterStep1(int *g_idata, int *g_odata, int size) {
	
	__shared__ int temp[blockSize];
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	temp[threadIdx.x] = g_idata[index];
	__syncthreads();
	if(index < size){
		if(temp[threadIdx.x] == 0)
			g_odata[index] = 0;
		else 
			g_odata[index] = 1;

		__syncthreads();
	}
}

__global__ void parallelScatterStep2(int *g_idata, int *g_odata, int size) {
	
	__shared__ int temp[blockSize];
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
		
	temp[threadIdx.x] = g_idata[index];
	__syncthreads();

	//if(temp[threadIdx.x] != 0){
	//	g_odata[postScanData[index - 1]] = temp[threadIdx.x];
	//}

	if(threadIdx.x == 0){
		if(index == 0)
			g_odata[index] = 0;
		else
			g_odata[index] = g_idata[index - 1];
	}
	else
		g_odata[index] = temp[threadIdx.x - 1];

	

}

__global__ void parallelScatterStep3(int *g_idata, int *intermediate, int *g_odata, int size) {
	__shared__ int temp[blockSize];
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
		
	temp[threadIdx.x] = g_idata[index];
	__syncthreads();	
	
	if(temp[threadIdx.x] != 0){
		int newIndex = intermediate[index];
	 	g_odata[newIndex] = temp[threadIdx.x];
	}

	__syncthreads();
}




void naiveParallelScan(const int *a, int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *temp = 0;

	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
	hipMalloc((void**)&temp, size * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

	//clock_t begin = clock();

	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord( start, 0);


	naiveParallelScan <<<1, size>>>(dev_a, dev_b, temp, size);

	// stop the timer
	hipEventRecord( stop, 0);
    hipEventSynchronize( stop );
	float time = 0.0f;
    hipEventElapsedTime( &time, start, stop);


	printf("Elapsed Time For GPU: %.4f \n", time);

	// Check for any errors launching the kernel
    hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
    hipMemcpy(b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost);
}

void shareMemoryParallelScan(const int *a, int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *temp = 0;


	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));


	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord( start, 0);

	sharedMemoryParallelScan <<<1, size>>>(dev_a, dev_b, size);

	// stop the timer
	hipEventRecord( stop, 0);
    hipEventSynchronize( stop );
	float time = 0.0f;
    hipEventElapsedTime( &time, start, stop);
	printf("Elapsed Time For GPU with Share memory: %.4f \n", time);

	// Check for any errors launching the kernel
    hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
    hipMemcpy(b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost);
}

void shareMemoryParallelScanArbitraryLength(const int *a, int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;


	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

	dim3 fullBlocksPerGrid((int)ceil(float(size)/float(blockSize)));
	
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord( start, 0);



	for(int offset = 1; offset <= size; offset *= 2){
		sharedMemoryParallelScanArbritraryLength2 <<<fullBlocksPerGrid, blockSize>>>(dev_a, dev_b, offset, size);
		hipMemcpy(dev_a, dev_b, size * sizeof(int), hipMemcpyDeviceToDevice);
		//dev_a = dev_b;
	}


	// stop the timer
	hipEventRecord( stop, 0);
    hipEventSynchronize( stop );
	float time = 0.0f;
    hipEventElapsedTime( &time, start, stop);
	printf("Elapsed Time For GPU with Share memory arbitrary length: %.4f \n", time);

	// Check for any errors launching the kernel
    hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
    hipMemcpy(b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost);
}


void parallelScatter(const int *a, int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *tempArrPreScan = 0;
	int *tempArrPostScan = 0;

	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));
	hipMalloc((void**)&tempArrPreScan, size * sizeof(int));
	hipMalloc((void**)&tempArrPostScan, size * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

	dim3 fullBlocksPerGrid((int)ceil(float(size)/float(blockSize)));
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord( start, 0);

	parallelScatterStep1<<<fullBlocksPerGrid, blockSize>>>(dev_a, tempArrPreScan, size);
	for(int offset = 1; offset <= size; offset *= 2){
		sharedMemoryParallelScanArbritraryLength2 <<<fullBlocksPerGrid, blockSize>>>(tempArrPreScan, tempArrPostScan, offset, size);
		hipMemcpy(tempArrPreScan, tempArrPostScan, size * sizeof(int), hipMemcpyDeviceToDevice);
	}


	parallelScatterStep2 <<<fullBlocksPerGrid, blockSize>>>(tempArrPreScan, tempArrPostScan, size);
	parallelScatterStep3 <<<fullBlocksPerGrid, blockSize>>>(dev_a, tempArrPostScan, dev_b, size);

	// stop the timer
	hipEventRecord( stop, 0);
    hipEventSynchronize( stop );
	float time = 0.0f;
    hipEventElapsedTime( &time, start, stop);

	// Check for any errors launching the kernel
    hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
    hipMemcpy(b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost);
}


