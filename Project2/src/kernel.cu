#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
//#include "glm/glm.hpp"
#include "utilities.h"
#include "kernel.h"


//#include "cuPrintf.cu"


//GLOBALS
dim3 threadsPerBlock(blockSize);

int numObjects;


void checkCUDAError(const char *msg, int line = -1)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        if( line >= 0 ) {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
        exit(EXIT_FAILURE); 
    }
} 


__global__ void naiveParallelScan (const int *a, int *b, int *temp, int size)
{
	int index = threadIdx.x;

	if(index < size){
		temp[index] = a[index];
		__syncthreads();

		for(int offset = 1; offset <= size; offset *= 2){

			if(index >= offset){
				b[index] = temp[index - offset] + temp[index];
			}
			else{
				b[index] = temp[index];
			}
			temp[index] = b[index];
			__syncthreads();
		}
	}
}


__global__ void sharedMemoryParallelScan (const int *a, int *b, int size){
	__shared__ int temp[blockSize];
	__shared__ int temp2[blockSize];

	int index = threadIdx.x;
	if(index < size )
	{
		temp[index] = a[index];
		__syncthreads();

		for(int offset = 1; offset <= size; offset *= 2){

			if(index >= offset){
				temp2[index] = temp[index - offset] + temp[index];
			}
			else{
				temp2[index] = temp[index];
			}

			temp[index] = temp2[index];
			__syncthreads();
		}
		b[index] = temp[index];
	}
}

__global__ void sharedMemoryParallelScanArbritraryLengthStep1 (const int *a, int *b, int size){
	__shared__ int temp[blockSize];
	__shared__ int temp2[blockSize];

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;


	temp[threadIdx.x] = a[index];
	__syncthreads();

	for(int offset = 1; offset <= blockSize; offset *= 2){

		if(threadIdx.x >= offset){
			temp2[threadIdx.x] = temp[threadIdx.x - offset] + temp[threadIdx.x];
		}
		else{
			temp2[threadIdx.x] = temp[threadIdx.x];
		}

		temp[threadIdx.x] = temp2[threadIdx.x];
		__syncthreads();
	}

	__syncthreads();

	b[index] = temp2[threadIdx.x];
}

__global__ void sharedMemoryParallelScanArbritraryLengthStep2 (const int *a, int *b, int size){
	__shared__ int temp[blockSize];

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	temp[threadIdx.x] = a[index];
	__syncthreads();

	for(int b = 0; b < blockIdx.x ; ++b){
		temp[threadIdx.x] += a[ (b + 1) * blockSize - 1];
	}
	b[index] = temp[threadIdx.x];
}


__global__ void scan(float *g_odata, float *g_idata, int n) 
{ 
	 extern __shared__ float temp[]; // allocated on invocation 
	 int thid = threadIdx.x; 
	 int pout = 0, pin = 1; 
	 // load input into shared memory. 
	 // This is exclusive scan, so shift right by one and set first elt to 0 
	 temp[pout*n + thid] = (thid > 0) ? g_idata[thid-1] : 0; 
	 __syncthreads(); 
	 for (int offset = 1; offset < n; offset *= 2) 
	 { 
		 pout = 1 - pout; // swap double buffer indices 
		 pin = 1 - pout; 
		 if (thid >= offset) 
			temp[pout*n+thid] += temp[pin*n+thid - offset]; 
		 else 
			temp[pout*n+thid] = temp[pin*n+thid]; 
		 __syncthreads(); 
	 } 
	 g_odata[thid] = temp[pout*n + thid]; // write output 
} 

__global__ void parallelScatterStep1(const int *g_idata, int *g_odata, int size) {
	
	__shared__ int temp[blockSize];
	__shared__ int temp2[blockSize];
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	temp[threadIdx.x] = g_idata[index];
	__syncthreads();
	if(index < size){
		if(temp[threadIdx.x] == 0)
			temp2[threadIdx.x] = 0;
		else 
			temp2[threadIdx.x] = 1;

		__syncthreads();


		for(int offset = 1; offset <= blockSize; offset *= 2){

			if(threadIdx.x >= offset){
				temp[threadIdx.x] = temp2[threadIdx.x] + temp2[threadIdx.x - offset];
			}
			else{
				temp[threadIdx.x] = temp2[threadIdx.x];
			}


			temp2[threadIdx.x] = temp[threadIdx.x];
			__syncthreads();
		}

		g_odata[index] = temp2[threadIdx.x];
	}
}

__global__ void parallelScatterStep2(const int *g_idata, int *g_odata, int size) {
	
	__shared__ int temp[blockSize];
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
		
	temp[threadIdx.x] = g_idata[index];
	__syncthreads();


	if(threadIdx.x == 0){
		if(index == 0)
			g_odata[index] = 0;
		else
			g_odata[index] = g_idata[index - 1];
	}
	else
		g_odata[index] = temp[threadIdx.x - 1];

}

__global__ void parallelScatterStep3(const int *g_idata, int *intermediate, int *g_odata, int size) {
	__shared__ int temp[blockSize];
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
		
	temp[threadIdx.x] = g_idata[index];
	__syncthreads();	
	
	if(temp[threadIdx.x] != 0){
		int newIndex = intermediate[index];
	 	g_odata[newIndex] = temp[threadIdx.x];
	}

	//if(g_idata[index] != 0){
	//	int newIndex = intermediate[index];
	// 	g_odata[newIndex] = g_idata[index];
	//}

	//__syncthreads();
}




void naiveParallelScan(const int *a, int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *temp = 0;

	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
	hipMalloc((void**)&temp, size * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);



	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord( start, 0);

	for(int iter = 0; iter < iterNum; ++iter){
		naiveParallelScan <<<1, size>>>(dev_a, dev_b, temp, size);
	}
	// stop the timer
	hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
	float time = 0.0f;
    hipEventElapsedTime( &time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Elapsed Time For GPU Naive Parallel Scan \n(Array length: %d, Iterate times: %d): %.4f ms \n", size, iterNum, time);

	// Check for any errors launching the kernel
    hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish
	hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
    hipMemcpy(b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost);
}

void shareMemoryParallelScan(const int *a, int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *temp = 0;


	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));


	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord( start, 0);

	for(int iter = 0; iter < iterNum; ++iter){
		sharedMemoryParallelScan <<<1, size>>>(dev_a, dev_b, size);
	}
	// stop the timer
	hipEventRecord( stop, 0);
    hipEventSynchronize( stop );
	float time = 0.0f;
    hipEventElapsedTime( &time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Elapsed Time For GPU Parallel Scan with Share memory \n(Array length: %d, Iterate times: %d): %.4f ms \n", size, iterNum, time);

	// Check for any errors launching the kernel
    hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
    hipMemcpy(b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost);
}

void shareMemoryParallelScanArbitraryLength(const int *a, int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *temp = 0;

	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
	hipMalloc((void**)&temp, size * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

	dim3 fullBlocksPerGrid((int)ceil(float(size)/float(blockSize)));
	
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord( start, 0);

	for(int iter = 0; iter < iterNum; ++iter){
		sharedMemoryParallelScanArbritraryLengthStep1 <<<fullBlocksPerGrid, blockSize>>>(dev_a, temp, size);
		sharedMemoryParallelScanArbritraryLengthStep2 <<<fullBlocksPerGrid, blockSize>>>(temp, dev_b, size);
	}

	// stop the timer
	hipEventRecord( stop, 0);
    hipEventSynchronize( stop );
	float time = 0.0f;
    hipEventElapsedTime( &time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Elapsed Time For GPU Parallel Scan with Share memory arbitrary length \n(Array length: %d, Iterate times: %d): %.4f ms \n", size, iterNum, time);

	// Check for any errors launching the kernel
    hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
    hipMemcpy(b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost);
}

void parallelScatter(const int *a, int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *tempArrPreScan = 0;
	int *tempArrPostScan = 0;
	int *intermediate = 0;

	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));
	hipMalloc((void**)&tempArrPreScan, size * sizeof(int));
	hipMalloc((void**)&tempArrPostScan, size * sizeof(int));
	hipMalloc((void**)&intermediate, size * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

	dim3 fullBlocksPerGrid((int)ceil(float(size)/float(blockSize)));
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord( start, 0);

	for(int iter = 0; iter < iterNum; ++iter){
		parallelScatterStep1<<<fullBlocksPerGrid, blockSize>>>(dev_a, tempArrPreScan, size);
		sharedMemoryParallelScanArbritraryLengthStep2 <<<fullBlocksPerGrid, blockSize>>>(tempArrPreScan, tempArrPostScan, size);
		parallelScatterStep2 <<<fullBlocksPerGrid, blockSize>>>(tempArrPostScan, intermediate, size);
		parallelScatterStep3 <<<fullBlocksPerGrid, blockSize>>>(dev_a, intermediate, dev_b, size);
	}


	// stop the timer
	hipEventRecord( stop, 0);
    hipEventSynchronize( stop );
	float time = 0.0f;
    hipEventElapsedTime( &time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Elapsed Time For GPU Scatter \n(Array length: %d, Iterate times: %d): %.4f ms \n", size, iterNum, time);

	// Check for any errors launching the kernel
    hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
    hipMemcpy(b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost);
}


