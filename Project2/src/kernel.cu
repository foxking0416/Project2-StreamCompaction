#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
//#include "glm/glm.hpp"
#include "utilities.h"
#include "kernel.h"

//#include "cuPrintf.cu"


//GLOBALS
dim3 threadsPerBlock(blockSize);

int numObjects;


void checkCUDAError(const char *msg, int line = -1)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        if( line >= 0 ) {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
        exit(EXIT_FAILURE); 
    }
} 



__global__ void naiveParallelScan (const int *a, int *b, int size, int cut)
{
	int i = threadIdx.x;


	if(i >= cut){
		b[i] = a[i - cut] + a[i];
	}
	else{
		b[i] = a[i];
	}

}

__global__ void sharedMemoryParallelScan (const int *a, int *b, int size, int cut){
	__shared__ int temp[blockSize];
	int index = threadIdx.x;

	temp[index] = a[index];
	__syncthreads();

	if(index >= cut){
		b[index] = temp[index - cut] + temp[index];
	}
	else{
		b[index] = temp[index];
	}
	__syncthreads();
}

__global__ void sharedMemoryParallelScanArbritraryLength (const int *a, int *b, int size, int cut){
	__shared__ int temp[blockSize];
	//__shared__ int temp2[blockSize];

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;


	temp[threadIdx.x] = a[index];
	__syncthreads();

	if(index >= cut){
		b[index] = a[index - cut] + temp[threadIdx.x];
	}
	else{
		b[index] = temp[threadIdx.x];
	}
	__syncthreads();
}

__global__ void scan(float *g_odata, float *g_idata, int n) 
{ 
	 extern __shared__ float temp[]; // allocated on invocation 
	 int thid = threadIdx.x; 
	 int pout = 0, pin = 1; 
	 // load input into shared memory. 
	 // This is exclusive scan, so shift right by one and set first elt to 0 
	 temp[pout*n + thid] = (thid > 0) ? g_idata[thid-1] : 0; 
	 __syncthreads(); 
	 for (int offset = 1; offset < n; offset *= 2) 
	 { 
		 pout = 1 - pout; // swap double buffer indices 
		 pin = 1 - pout; 
		 if (thid >= offset) 
			temp[pout*n+thid] += temp[pin*n+thid - offset]; 
		 else 
			temp[pout*n+thid] = temp[pin*n+thid]; 
		 __syncthreads(); 
	 } 
	 g_odata[thid] = temp[pout*n + thid]; // write output 
} 

__global__ void parallelScatter(int *g_idata, int *g_odata, int n) {
	int i = threadIdx.x;


	if(g_idata[i] == 0)
		g_odata[i] = 0;
	else 
		g_odata[i] = 1;
}

__global__ void parallelScatter2(int *g_idata, int *postScanData, int *g_odata, int n) {
	int i = threadIdx.x;

	if(g_idata[i] != 0){
		g_odata[postScanData[i-1]] = g_idata[i];
	}
}


void naiveParallelScanWithCuda(const int *a, int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *temp = 0;

	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
	hipMalloc((void**)&temp, size * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

	//clock_t begin = clock();

	//Naive Parallel Scan

	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord( start, 0);

	for(int d = 1; d <= (int)ceil(log2((float) size)); ++d){
		int cut = (int)pow((float)2, (d - 1));
		naiveParallelScan <<<1, size>>>(dev_a, dev_b, size, cut);
		dev_a = dev_b;
	}
	
	// stop the timer
	hipEventRecord( stop, 0);
    hipEventSynchronize( stop );
	float time = 0.0f;
    hipEventElapsedTime( &time, start, stop);


	printf("Elapsed Time For GPU: %.4f \n", time);

	// Check for any errors launching the kernel
    hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
    hipMemcpy(b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost);
}

void shareMemoryParallelScanWithCuda(const int *a, int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;

	//hipError_t cudaStatus;

	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord( start, 0);

	//Share memory
	for(int d = 1; d <= (int)ceil(log2((float) size)); ++d){
		int cut = (int)pow((float)2, (d - 1));
		sharedMemoryParallelScan <<<1, size>>>(dev_a, dev_b, size, cut);
		dev_a = dev_b;
	}

	// stop the timer
	hipEventRecord( stop, 0);
    hipEventSynchronize( stop );
	float time = 0.0f;
    hipEventElapsedTime( &time, start, stop);
	printf("Elapsed Time For GPU with Share memory: %.4f \n", time);

	// Check for any errors launching the kernel
    hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
    hipMemcpy(b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost);
}

void shareMemoryParallelScanArbitraryLengthWithCuda(const int *a, int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;

	//hipError_t cudaStatus;

	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

	dim3 fullBlocksPerGrid((int)ceil(float(size)/float(blockSize)));
	//Share memory
	for(int d = 1; d <= (int)ceil(log2((float) size)); ++d){
		int cut = (int)pow((float)2, (d - 1));
		sharedMemoryParallelScanArbritraryLength <<<fullBlocksPerGrid, blockSize>>>(dev_a, dev_b, size, cut);
		dev_a = dev_b;
	}

	// Check for any errors launching the kernel
    hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
    hipMemcpy(b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost);
}


void parallelScatterWithCuda(const int *a, int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *tempArrPreScan = 0;
	int *tempArrPostScan = 0;

	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));
	hipMalloc((void**)&tempArrPreScan, size * sizeof(int));
	hipMalloc((void**)&tempArrPostScan, size * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

	parallelScatter<<<1, size>>>(dev_a, tempArrPreScan, size);
	for(int d = 1; d <= (int)ceil(log2((float) size)); ++d){
		int cut = (int)pow((float)2, (d - 1));
		sharedMemoryParallelScan <<<1, size>>>(tempArrPreScan, tempArrPostScan, size, cut);
		tempArrPreScan = tempArrPostScan;
	}

	parallelScatter2 <<<1, size>>>(dev_a, tempArrPostScan, dev_b, size);

	// Check for any errors launching the kernel
    hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
    hipMemcpy(b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost);
}



